#include "hip/hip_runtime.h"
//
// Created by developer on 5/24/20.
//

#include "saxpy.cuh"

__global__
void run(int n, float a, float *d_x, float *d_y) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) d_y[i] = a * d_x[i] + d_y[i];
}

void saxpy(int N, float a, float *x, float *y)
{
    float *d_x, *d_y;

    hipMalloc(&d_x, N*sizeof(float));
    hipMalloc(&d_y, N*sizeof(float));

    hipMemcpy(d_x, x, N*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, N*sizeof(float), hipMemcpyHostToDevice);

    // Perform SAXPY on N elements
    run<<<(N+255)/256, 256>>>(N, a, d_x, d_y);

    hipMemcpy(y, d_y, N*sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_x);
    hipFree(d_y);
}